
#include <hip/hip_runtime.h>
#include <iostream>

extern "C" __global__ void add(const float* a, const float *b, float *c, int N) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N) {
	  c[idx] = a[idx] + b[idx];
  }
}

#if 1
int main(void) {
  float a[10], b[10], c[10];
  // init the cpu buffer
  for (int i = 0; i < 10; ++i) {
    a[i] = i;
    b[i] = i * 10;
  }
  // copy cpu buffer to gpu buffer
  float *agpu, *bgpu, *cgpu;
  hipMalloc((void **) &agpu, sizeof(a));
  hipMalloc((void **) &bgpu, sizeof(b));
  hipMalloc((void **) &cgpu, sizeof(c));
  hipMemcpy(agpu, a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpy(bgpu, b, sizeof(b), hipMemcpyHostToDevice);

  add<<<10, 1>>>(agpu, bgpu, cgpu, 10);
  hipMemcpy(c, cgpu, sizeof(c), hipMemcpyDeviceToHost);
  for (int i = 0; i < 10; ++i) {
    std::cout << c[i] << std::endl;
  }
  return 0;
}
#endif
